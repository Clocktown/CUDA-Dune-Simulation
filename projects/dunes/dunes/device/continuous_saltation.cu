#include "hip/hip_runtime.h"
#include "continuous_saltation.cuh"
#include "constants.cuh"
#include "grid.cuh"
#include <dunes/core/simulation_parameters.hpp>
#include <dunes/core/launch_parameters.hpp>
#include <sthe/device/vector_extension.cuh>
#include <sthe/config/debug.hpp>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <cstdio>

namespace dunes
{

__global__ void setupContinuousSaltationKernel(Array2D<float2> t_terrainArray, const Array2D<float2> t_windArray, Array2D<float4> t_resistanceArray, Buffer<float> t_slabBuffer, Buffer<float> t_advectedSlabBuffer)
{
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{
			float2 terrain{ t_terrainArray.read(cell) };

			const float2 windVelocity{ t_windArray.read(cell) };
			const float windSpeed{ length(windVelocity) };

			const float4 resistance{ t_resistanceArray.read(cell) };
			const float saltationScale{ (1.0f - resistance.x) * (1.0f - resistance.y) * (resistance.w > 0.0f ? 0.5f : 1.0f) };

			//const float scale{ windSpeed * c_parameters.deltaTime };

			const float saltation{ fminf(c_parameters.saltationStrength * saltationScale + (resistance.w < 0.0f ? -resistance.w : 0.0f), terrain.y) };

			terrain.y -= saltation;
			t_terrainArray.write(cell, terrain);

			const int cellIndex{ getCellIndex(cell) };
			const float slab{ saltation };

			t_slabBuffer[cellIndex] += slab;
			t_advectedSlabBuffer[cellIndex] = 0.0f;
		}
	}
}

__global__ void continuousSaltationKernel(const Array2D<float2> t_windArray, Buffer<float> t_slabBuffer, Buffer<float> t_advectedSlabBuffer)
{
	const int2 cell{ getGlobalIndex2D() };

	if (isOutside(cell))
	{
		return;
	}

	const int cellIndex{ getCellIndex(cell) };
	const float slab{ t_slabBuffer[cellIndex] };

	const float2 windVelocity{ t_windArray.read(cell) };

	const float2 position{ make_float2(cell) };

	if (slab > 0.0f)
	{
		const float2 nextPosition{ position + windVelocity * c_parameters.rGridScale * c_parameters.deltaTime };
		//const int2 nextCell{ getNearestCell(nextPosition) };
		const int2 nextCell{ make_int2(nextPosition) };

		for (int x{ nextCell.x }; x <= nextCell.x + 1; ++x)
		{
			const float u{ 1.0f - abs(static_cast<float>(x) - nextPosition.x) };

			for (int y{ nextCell.y }; y <= nextCell.y + 1; ++y)
			{
				const float v{ 1.0f - abs(static_cast<float>(y) - nextPosition.y) };
				const float weight{ u * v };
		
				if (weight > 0.0f)
				{
					atomicAdd(t_advectedSlabBuffer + getCellIndex(getWrappedCell(nextCell)), weight * slab);
				}
			}
		}
		
		//atomicAdd(t_advectedSlabBuffer + getCellIndex(getWrappedCell(nextCell)), slab);
	}
}

__global__ void finishContinuousSaltationKernel(Array2D<float2> t_terrainArray, const Array2D<float2> t_windArray, const Array2D<float4> t_resistanceArray, Buffer<float> t_slabBuffer, Buffer<float> t_advectedSlabBuffer)
{
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{
			const int cellIndex{ getCellIndex(cell) };

			float2 terrain{ t_terrainArray.read(cell) };
			const float slab{ t_advectedSlabBuffer[cellIndex] };

			const float windSpeed{ length(t_windArray.read(cell)) };
			
			const float4 resistance{ t_resistanceArray.read(cell) };
			const float saltationScale{ (1.0f - resistance.x) * (1.0f - resistance.y) };
			const float abrasionScale{ saltationScale * (1.0f - resistance.z) };
			const float vegetationFactor = (terrain.y > 0.0f ? 0.4f : 0.6f);
			const float depositionProbability = fminf(fmaxf(fmaxf(resistance.x,
				(1.0f - vegetationFactor) + resistance.y * vegetationFactor), resistance.w), resistance.w < 0.f ? 0.f : 1.f);


			const float new_slab = slab * (1.f - depositionProbability);
			//if (new_slab > 0.0f) {
				const float scale{ slab + c_parameters.windOnlyAbrasionAmount }; // 0.0001 is the amount of abrasion that happens with purely wind, no sand 
				float abrasion{ terrain.y < c_parameters.abrasionThreshold ?
				c_parameters.abrasionStrength * abrasionScale * windSpeed * c_parameters.deltaTime *
				clamp(1.0f - terrain.y / c_parameters.abrasionThreshold, 0.f, 1.f) * scale : 0.0f };
				//abrasion = fminf(abrasion, terrain.x);

				terrain.y += abrasion;
				terrain.x -= abrasion;
			//}
			terrain.y += slab * depositionProbability;
			t_terrainArray.write(cell, terrain);
			t_slabBuffer[cellIndex] = slab * (1.f - depositionProbability); // write updated advectedSlabBuffer back to slabBuffer (ping-pong)
			t_advectedSlabBuffer[cellIndex] = slab * (1.f - resistance.y); // Used in Reptation as slabBuffer
		}
	}
}

void continuousSaltation(const LaunchParameters& t_launchParameters)
{
	setupContinuousSaltationKernel<<<t_launchParameters.optimalGridSize2D, t_launchParameters.optimalBlockSize2D>>>(t_launchParameters.terrainArray, t_launchParameters.windArray, t_launchParameters.resistanceArray, t_launchParameters.slabBuffer, t_launchParameters.tmpBuffer);
	continuousSaltationKernel<<<t_launchParameters.gridSize2D, t_launchParameters.blockSize2D>>>(t_launchParameters.windArray, t_launchParameters.slabBuffer, t_launchParameters.tmpBuffer);
	finishContinuousSaltationKernel<<<t_launchParameters.optimalGridSize2D, t_launchParameters.optimalBlockSize2D>>>(t_launchParameters.terrainArray, t_launchParameters.windArray, t_launchParameters.resistanceArray, t_launchParameters.slabBuffer, t_launchParameters.tmpBuffer);
}

}
