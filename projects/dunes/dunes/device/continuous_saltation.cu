#include "hip/hip_runtime.h"
#include "continuous_saltation.cuh"
#include "constants.cuh"
#include "grid.cuh"
#include <dunes/core/simulation_parameters.hpp>
#include <dunes/core/launch_parameters.hpp>
#include <sthe/device/vector_extension.cuh>
#include <sthe/config/debug.hpp>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <cstdio>

namespace dunes
{

__global__ void setupContinuousSaltationKernel(Array2D<float2> t_terrainArray, const Array2D<float2> t_windArray, Array2D<float4> t_resistanceArray, Buffer<float> t_slabBuffer, Buffer<float> t_advectedSlabBuffer)
{
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{
			float2 terrain{ t_terrainArray.read(cell) };

			const float2 windVelocity{ t_windArray.read(cell) };
			const float windSpeed{ length(windVelocity) };

			const float4 resistance{ t_resistanceArray.read(cell) };
			const float saltationResistance{ (1.0f - resistance.x) * (1.0f - resistance.y) };
			const float abrasionResistance{ saltationResistance * (1.0f - resistance.z) };

			const float scale{ windSpeed * c_parameters.deltaTime };
			const float abrasion{ fminf(terrain.y < c_parameters.abrasionThreshold ?
										c_parameters.abrasionStrength * abrasionResistance *
										(1.0f - terrain.y / c_parameters.abrasionThreshold) * scale : 0.0f, terrain.x) };

			const float saltation{ fminf(c_parameters.saltationStrength * saltationResistance * scale, terrain.y) };

			terrain.x -= abrasion;
			terrain.y -= saltation;
			t_terrainArray.write(cell, terrain);

			const int cellIndex{ getCellIndex(cell) };
			const float slab{ saltation + abrasion };

			t_slabBuffer[cellIndex] += slab;
			t_advectedSlabBuffer[cellIndex] = 0.0f;
		}
	}
}

__global__ void continuousSaltationKernel(const Array2D<float2> t_windArray, Buffer<float> t_slabBuffer, Buffer<float> t_advectedSlabBuffer)
{
	const int2 cell{ getGlobalIndex2D() };

	if (isOutside(cell))
	{
		return;
	}

	const int cellIndex{ getCellIndex(cell) };
	const float slab{ t_slabBuffer[cellIndex] };

	const float2 windVelocity{t_windArray.read(cell) };

	const float2 position{ make_float2(cell) };

	if (slab > 0.0f)
	{
		const float2 nextPosition{ position + windVelocity * c_parameters.rGridScale * c_parameters.deltaTime };
		const int2 nextCell{ make_int2(nextPosition) };

		for (int x{ nextCell.x }; x <= nextCell.x + 1; ++x)
		{
			const float u{ 1.0f - abs(static_cast<float>(x) - nextPosition.x) };

			for (int y{ nextCell.y }; y <= nextCell.y + 1; ++y)
			{
				const float v{ 1.0f - abs(static_cast<float>(y) - nextPosition.y) };
				const float weight{ u * v };

				if (weight > 0.0f)
				{
					atomicAdd(t_advectedSlabBuffer + getCellIndex(getWrappedCell(int2{ x, y })), weight * slab);
				}
			}
		}
	}
}

__global__ void finishContinuousSaltationKernel(Array2D<float2> t_terrainArray, const Array2D<float4> t_resistanceArray, Buffer<float> t_slabBuffer, const Buffer<float> t_advectedSlabBuffer)
{
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{
			const int cellIndex{ getCellIndex(cell) };

			float2 terrain{ t_terrainArray.read(cell) };
			const float slab{ t_advectedSlabBuffer[cellIndex] };

			const float4 resistance{ t_resistanceArray.read(cell) };
			const float vegetationFactor = (terrain.y > 0.0f ? 0.4f : 0.6f);
			const float depositionProbability = fmaxf(resistance.x,
				(1.0f - vegetationFactor) + resistance.y * vegetationFactor);

			terrain.y += slab * depositionProbability;

			t_terrainArray.write(cell, terrain);
			t_slabBuffer[cellIndex] = slab * (1.f - depositionProbability);
			t_advectedSlabBuffer[cellIndex] = slab * (1.f - resistance.y);
		}
	}
}

void continuousSaltation(const LaunchParameters& t_launchParameters)
{
	setupContinuousSaltationKernel<<<t_launchParameters.optimalGridSize2D, t_launchParameters.optimalBlockSize2D>>>(t_launchParameters.terrainArray, t_launchParameters.windArray, t_launchParameters.resistanceArray, t_launchParameters.slabBuffer, t_launchParameters.tmpBuffer);
	continuousSaltationKernel<<<t_launchParameters.gridSize2D, t_launchParameters.blockSize2D>>>(t_launchParameters.windArray, t_launchParameters.slabBuffer, t_launchParameters.tmpBuffer);
	finishContinuousSaltationKernel<<<t_launchParameters.optimalGridSize2D, t_launchParameters.optimalBlockSize2D>>>(t_launchParameters.terrainArray, t_launchParameters.resistanceArray, t_launchParameters.slabBuffer, t_launchParameters.tmpBuffer);
}

}
