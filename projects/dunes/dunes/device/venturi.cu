#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "constants.cuh"
#include "grid.cuh"
#include <dunes/core/simulation_parameters.hpp>
#include <dunes/core/launch_parameters.hpp>
#include <sthe/device/vector_extension.cuh>

namespace dunes
{

__global__ void venturiKernel(Array2D<float2> t_terrainArray, Array2D<float2> t_windArray)
{
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{

			const float2 terrain{ t_terrainArray.read(cell) };
			const float height{ terrain.x + terrain.y };

			const float venturiScale{ (1.0f + c_parameters.venturiStrength * height) };
			float2 windVelocity{ venturiScale * c_parameters.windSpeed * c_parameters.windDirection };
			windVelocity *= sinf(height);
			t_windArray.write(cell, windVelocity);
		}
	}
}

void venturi(const LaunchParameters& t_launchParameters)
{
	venturiKernel<<<t_launchParameters.optimalGridSize2D, t_launchParameters.optimalBlockSize2D>>>(t_launchParameters.terrainArray, t_launchParameters.windArray);
}

}
