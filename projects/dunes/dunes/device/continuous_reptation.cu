#include "hip/hip_runtime.h"
#include "continuous_reptation.cuh"
#include "constants.cuh"
#include "grid.cuh"
#include <dunes/core/simulation_parameters.hpp>
#include <dunes/core/launch_parameters.hpp>
#include <sthe/device/vector_extension.cuh>
#include <sthe/config/debug.hpp>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <cstdio>

namespace dunes
{

__global__ void setupContinuousReptationKernel(Buffer<float> t_reptationBuffer)
{
	const int stride{ getGridStride1D() };

	for (int cellIndex{ getGlobalIndex1D() }; cellIndex < c_parameters.cellCount; cellIndex += stride)
	{
		t_reptationBuffer[cellIndex] = 0.0f;
	}
}

__global__ void continuousReptationKernel(const Array2D<float2> t_terrainArray, Buffer<float> t_slabBuffer, Buffer<float> t_reptationBuffer)
{
	const int2 cell{ getGlobalIndex2D() };

	if (isOutside(cell))
	{
		return;
	}

	const int cellIndex{ getCellIndex(cell) };
	const float2 terrain{ t_terrainArray.read(cell) };
	const float height{ terrain.x + terrain.y };

	float slab{ t_slabBuffer[cellIndex] };

	float change{ 0.0f };

	for (int i{ 0 }; i < 8; ++i)
	{
		const int2 nextCell{ getWrappedCell(cell + c_offsets[i]) };
		const float nextSlab{ t_slabBuffer[getCellIndex(nextCell)] };

		const float2 nextTerrain{ t_terrainArray.read(nextCell) };
		const float nextHeight{ nextTerrain.x + nextTerrain.y };

		const float heightDifference{ (nextHeight - height) * c_parameters.rGridScale * c_rDistances[i]};
		const float heightScale = 1.f;// fmaxf(c_parameters.avalancheAngle - abs(heightDifference), 0.f) / c_parameters.avalancheAngle;

		// Enforce symmetric additive and subtractive changes, avoiding any atomics
		float step = fmaxf(0.5f * heightScale * (slab + nextSlab) * c_parameters.reptationStrength, 0.f);
        change += signbit(heightDifference) ? -fminf(step, terrain.y) : fminf(step, nextTerrain.y);
	}

	t_reptationBuffer[cellIndex] = change * 0.125;
}

__global__ void finishContinuousReptationKernel(Array2D<float2> t_terrainArray, Buffer<float> t_reptationBuffer)
{
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{
			const int cellIndex{ getCellIndex(cell) };

			float2 terrain{ t_terrainArray.read(cell) };
			terrain.y += t_reptationBuffer[getCellIndex(cell)];

			t_terrainArray.write(cell, terrain);
		}
	}
}

void continuousReptation(const LaunchParameters& t_launchParameters, const SimulationParameters& t_simulationParameters)
{
	if (t_simulationParameters.reptationStrength > 0.f) {
		Buffer<float> reptationBuffer{ t_launchParameters.tmpBuffer + t_simulationParameters.cellCount };

		//if (t_simulationParameters.timestep == 0) {
		//	setupContinuousReptationKernel << <t_launchParameters.optimalGridSize1D, t_launchParameters.optimalBlockSize1D >> > (reptationBuffer);
		//}
		continuousReptationKernel << <t_launchParameters.gridSize2D, t_launchParameters.blockSize2D >> > (t_launchParameters.terrainArray, t_launchParameters.tmpBuffer, reptationBuffer);
		finishContinuousReptationKernel << <t_launchParameters.optimalGridSize2D, t_launchParameters.optimalBlockSize2D >> > (t_launchParameters.terrainArray, reptationBuffer);
	}
}

}
