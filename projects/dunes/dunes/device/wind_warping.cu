#include "hip/hip_runtime.h"
#include "multigrid.cuh"
#include "constants.cuh"
#include "grid.cuh"
#include <dunes/core/simulation_parameters.hpp>
#include <dunes/core/launch_parameters.hpp>
#include <sthe/device/vector_extension.cuh>
#include <sthe/config/debug.hpp>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <cstdio>

#define RSQRT2Pi 0.3989422804f

namespace dunes
{

__forceinline__ __device__ float gaussian(const float t_x, const float t_sigma)
{
	const float rsigma{ 1.0f / t_sigma };
	return RSQRT2Pi * rsigma * expf(-0.5f * t_x * t_x * rsigma * rsigma);
}

__global__ void initializeWindWarpingKernel(WindWarping t_windWarping)
{
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	const int2 center{ c_parameters.gridSize / 2 };
	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{
			const int cellIndex{ getCellIndex(cell) };

			int2 fftshift{ 0, 0 };

			if (cell.x >= center.x)
			{
				fftshift.x = c_parameters.gridSize.x;
			}
			if (cell.y >= center.y)
			{
				fftshift.y = c_parameters.gridSize.y;
			}

			const float distance{ length(c_parameters.gridScale * (make_float2(cell  - fftshift) + 0.5))};
		
			for (int i{ 0 }; i < t_windWarping.count; ++i)
			{
				t_windWarping.gaussKernels[i][cellIndex] = hipComplex{ gaussian(distance, 0.5f * t_windWarping.radii[i]), 0.0f };
			}
		}
	}
}

__global__ void setupWindWarpingKernel(Array2D<float2> t_terrainArray, Buffer<hipComplex> t_heightBuffer)
{
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	const float2 center{ 0.5f * make_float2(c_parameters.gridSize) };
	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{
			const int cellIndex{ getCellIndex(cell) };
			const float2 terrain{ t_terrainArray.read(cell) };
			const float height{ terrain.x + terrain.y };

			t_heightBuffer[cellIndex] = hipComplex{ height, 0.0f };
		}
	}
}

__global__ void smoothTerrainsKernel(Buffer<hipComplex> t_heightBuffer, WindWarping t_windWarping)
{
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{
			const int cellIndex{ getCellIndex(cell) };
			const hipComplex height{ t_heightBuffer[cellIndex] };
			const float fftScale{ 1.0f / static_cast<float>(c_parameters.gridSize.x * c_parameters.gridSize.y) };

			for (int i{ 0 }; i < t_windWarping.count; ++i)
			{
				const hipComplex gauss = t_windWarping.gaussKernels[i][cellIndex];
				const hipComplex result = { // Complex Multiplication
					gauss.x * height.x - gauss.y * height.y, 
					gauss.x * height.y + gauss.y * height.x 
				};
				t_windWarping.smoothedHeights[i][cellIndex] = fftScale * result;
			}
		}
	}
}

__global__ void readBackSmoothTerrainKernel(Array2D<float2> t_terrainArray, WindWarping t_windWarping) {
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{
			const int cellIndex{ getCellIndex(cell) };
			
			t_terrainArray.write(cell, float2{ t_windWarping.smoothedHeights[0][cellIndex].x, 0.f });
		}
	}
}

__global__ void readGaussKernel(Array2D<float2> t_terrainArray, WindWarping t_windWarping) {
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{
			const int cellIndex{ getCellIndex(cell) };
			
			t_terrainArray.write(cell, 25000.f * float2{ t_windWarping.gaussKernels[0][cellIndex].x, 0.f });
		}
	}
}

__global__ void scaleGaussKernel(hipComplex* t_gauss, float scale) {
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{
			const int cellIndex{ getCellIndex(cell) };
			
			t_gauss[cellIndex].x *= scale;
		}
	}
}

__global__ void windWarpingKernel(Array2D<float2> t_windArray, WindWarping t_windWarping)
{
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{
			const int cellIndex{ getCellIndex(cell) };
			const float2 windVelocity{ t_windArray.read(cell) };
			const float windSpeed{ length(windVelocity) };
			//const float2 windDirection{ windVelocity / (windSpeed + 0.000001f) };
			
			float2 warpDirection{ 0.0f, 0.0f };
			float weight{ 0.0f };

			for (int i{ 0 }; i < t_windWarping.count; ++i)
			{
				const float smoothedHeights[4]{ t_windWarping.smoothedHeights[i][getCellIndex(getWrappedCell(cell + int2{ -1, 0 }))].x,
								                t_windWarping.smoothedHeights[i][getCellIndex(getWrappedCell(cell + int2{ 1, 0 }))].x,
								                t_windWarping.smoothedHeights[i][getCellIndex(getWrappedCell(cell + int2{ 0, -1 }))].x,
								                t_windWarping.smoothedHeights[i][getCellIndex(getWrappedCell(cell + int2{ 0, 1 }))].x };

				const float scale{ t_windWarping.i_divisor * 0.5f * c_parameters.rGridScale };
				const float2 gradient{ scale * (smoothedHeights[1] - smoothedHeights[0]),
								       scale * (smoothedHeights[3] - smoothedHeights[2]) };

				const float gradientLength{ length(gradient) };
				
				float2 orthogonalDirection{ -gradient.y, gradient.x };
				orthogonalDirection *= sign(dot(windVelocity, orthogonalDirection));
				
				float alpha{ fminf(gradientLength, 1.0f) }; 
			
				warpDirection += t_windWarping.strengths[i] * lerp(windVelocity, t_windWarping.gradientStrengths[i] * orthogonalDirection, alpha);
				weight += t_windWarping.strengths[i];
			}

			if (weight > 0.0f)
			{
				warpDirection /= weight;
			}

			warpDirection /= (length(warpDirection) + 0.000001f);
			t_windArray.write(cell, warpDirection * windSpeed);
		}
	}
}

__global__ void windWarpingKernelImproved(Array2D<float2> t_windArray, WindWarping t_windWarping)
{
	const int2 index{ getGlobalIndex2D() };
	const int2 stride{ getGridStride2D() };

	int2 cell;

	for (cell.x = index.x; cell.x < c_parameters.gridSize.x; cell.x += stride.x)
	{
		for (cell.y = index.y; cell.y < c_parameters.gridSize.y; cell.y += stride.y)
		{
			const int cellIndex{ getCellIndex(cell) };
			const float2 windVelocity{ t_windArray.read(cell) };
			const float windSpeed{ length(windVelocity) };
			//const float2 windDirection{ windVelocity / (windSpeed + 0.000001f) };
			
			float warpAngle{ 0.0f };
			float weight{ 0.0f };

			for (int i{ 0 }; i < t_windWarping.count; ++i)
			{
				const float smoothedHeights[4]{ t_windWarping.smoothedHeights[i][getCellIndex(getWrappedCell(cell + int2{ -1, 0 }))].x,
								                t_windWarping.smoothedHeights[i][getCellIndex(getWrappedCell(cell + int2{ 1, 0 }))].x,
								                t_windWarping.smoothedHeights[i][getCellIndex(getWrappedCell(cell + int2{ 0, -1 }))].x,
								                t_windWarping.smoothedHeights[i][getCellIndex(getWrappedCell(cell + int2{ 0, 1 }))].x };

				const float scale{ t_windWarping.gradientStrengths[i] * 0.5f * c_parameters.rGridScale };
				const float2 gradient{ scale * (smoothedHeights[1] - smoothedHeights[0]),
								       scale * (smoothedHeights[3] - smoothedHeights[2]) };

				const float gradientLength{ length(gradient) };
				
				float2 orthogonalDirection{ -gradient.y, gradient.x };
				orthogonalDirection *= sign(dot(windVelocity, orthogonalDirection));
				
				float alpha{ fminf(gradientLength / windSpeed, 1.0f) }; 
			
				warpAngle += t_windWarping.strengths[i] * alpha * signed_angle(windVelocity, orthogonalDirection);
				weight += t_windWarping.strengths[i];
			}

			if (weight > 0.0f)
			{
				warpAngle /= weight;
			}

			t_windArray.write(cell, rotate(windVelocity, warpAngle));
		}
	}
}

void initializeWindWarping(const LaunchParameters& t_launchParameters, const SimulationParameters& t_simulationParameters)
{
	initializeWindWarpingKernel << <t_launchParameters.optimalGridSize2D, t_launchParameters.optimalBlockSize2D >> > (t_launchParameters.windWarping);

	// Normalize Kernels
	for (int i = 0; i < t_launchParameters.windWarping.count; ++i) {
		float result = thrust::reduce(thrust::device, (float*)t_launchParameters.windWarping.gaussKernels[i], (float*)(t_launchParameters.windWarping.gaussKernels[i] + t_simulationParameters.cellCount));
		scaleGaussKernel << < t_launchParameters.optimalGridSize2D, t_launchParameters.optimalBlockSize2D >> > (t_launchParameters.windWarping.gaussKernels[i], 1.f / result);
	}

	for (int i{ 0 }; i < t_launchParameters.windWarping.count; ++i)
	{
		CUFFT_CHECK_ERROR(hipfftExecC2C(t_launchParameters.fftPlan, t_launchParameters.windWarping.gaussKernels[i], t_launchParameters.windWarping.gaussKernels[i], HIPFFT_FORWARD));
	}
}

void windWarping(const LaunchParameters& t_launchParameters)
{
	if (t_launchParameters.windWarpingMode == WindWarpingMode::Standard)
	{
		Buffer<hipComplex> heightBuffer{ reinterpret_cast<Buffer<hipComplex>>(t_launchParameters.tmpBuffer) };
	
	    setupWindWarpingKernel<<<t_launchParameters.optimalGridSize2D, t_launchParameters.optimalBlockSize2D>>>(t_launchParameters.terrainArray, heightBuffer);

	    CUFFT_CHECK_ERROR(hipfftExecC2C(t_launchParameters.fftPlan, heightBuffer, heightBuffer, HIPFFT_FORWARD));

	    smoothTerrainsKernel<<< t_launchParameters.optimalGridSize2D, t_launchParameters.optimalBlockSize2D>>>(heightBuffer, t_launchParameters.windWarping);

	    for (int i{ 0 }; i < t_launchParameters.windWarping.count; ++i)
	    {
		    CUFFT_CHECK_ERROR(hipfftExecC2C(t_launchParameters.fftPlan, t_launchParameters.windWarping.smoothedHeights[i], t_launchParameters.windWarping.smoothedHeights[i], HIPFFT_BACKWARD));
	    }

		//readBackSmoothTerrainKernel << < t_launchParameters.optimalGridSize2D, t_launchParameters.optimalBlockSize2D >> > (t_launchParameters.terrainArray, t_launchParameters.windWarping);

	    windWarpingKernel<<<t_launchParameters.optimalGridSize2D, t_launchParameters.optimalBlockSize2D>>>(t_launchParameters.windArray, t_launchParameters.windWarping);
	}
}

}
