#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "constants.cuh"
#include "grid.cuh"
#include "multigrid.cuh"
#include <sthe/config/debug.hpp>
#include <dunes/core/simulation_parameters.hpp>
#include <dunes/core/launch_parameters.hpp>
#include <sthe/device/vector_extension.cuh>
#include <thrust/execution_policy.h>
#include <thrust/transform_reduce.h>

namespace dunes {

	__global__ void initDivergencePressureKernel(const Array2D<float2> t_windArray, Buffer<float> t_divergenceBuffer, Buffer<float> t_pressureBuffer) {
		const int2 cell{ getGlobalIndex2D() };

		if (isOutside(cell))
		{
			return;
		}

		const int cellIndex{ getCellIndex(cell) };
		t_pressureBuffer[cellIndex] = 0.f;

		const float divergence = -0.5f * (
				(t_windArray.read(getWrappedCell(cell + c_offsets[0])).x - t_windArray.read(getWrappedCell(cell + c_offsets[4])).x) +
				(t_windArray.read(getWrappedCell(cell + c_offsets[2])).y - t_windArray.read(getWrappedCell(cell + c_offsets[6])).y)
			);

		t_divergenceBuffer[cellIndex] = divergence;
	}

	__global__ void projectKernel(const Array2D<float4> t_resistanceArray, const Buffer<float> t_divergenceBuffer, const Buffer<float> t_pressureABuffer, Buffer<float> t_pressureBBuffer) {
		const int2 cell{ getGlobalIndex2D() };

		if (isOutside(cell))
		{
			return;
		}

		const int cellIndex{ getCellIndex(cell) };

		float new_pressure = t_divergenceBuffer[cellIndex];
		for (int i = 0; i < 8; i += 2) {
			const int2 nextCell = getWrappedCell(cell + c_offsets[i]);
			const int nextCellIndex = getCellIndex(nextCell);

			new_pressure += t_pressureABuffer[nextCellIndex];
		}
		new_pressure *= 0.25f;
		//new_pressure *= (1.f - t_resistanceArray.read(cell).x);

		t_pressureBBuffer[cellIndex] = new_pressure;
	}

	__global__ void finalizeVelocities(Array2D<float4> t_resistanceArray, Array2D<float2> t_windArray, const Buffer<float> t_pressureBuffer) {
		const int2 cell{ getGlobalIndex2D() };

		if (isOutside(cell))
		{
			return;
		}

		const int cellIndex{ getCellIndex(cell) };

		float2 velocity = t_windArray.read(cell);
		float4 resistance = t_resistanceArray.read(cell);
		resistance.x = 0.0f;

		velocity.x -= 0.5f * (
				t_pressureBuffer[getCellIndex(getWrappedCell(cell + c_offsets[0]))] 
			-	t_pressureBuffer[getCellIndex(getWrappedCell(cell + c_offsets[4]))]
			);
		velocity.y -= 0.5f * (
				t_pressureBuffer[getCellIndex(getWrappedCell(cell + c_offsets[2]))] 
			-	t_pressureBuffer[getCellIndex(getWrappedCell(cell + c_offsets[6]))]
			);

		t_windArray.write(cell, velocity);
		//t_resistanceArray.write(cell, resistance);
	}

	__global__ void multiplyWindShadowKernel(Array2D<float2> t_windArray, Array2D<float4> t_resistanceArray) {
		const int2 cell{ getGlobalIndex2D() };

		if (isOutside(cell))
		{
			return;
		}

		const int cellIndex{ getCellIndex(cell) };

		float2 velocity = t_windArray.read(cell) * (1.f - t_resistanceArray.read(cell).x);
		
		t_windArray.write(cell, velocity);
	}

	__global__ void setupProjection(const Array2D<float2> t_windArray, Array2D<float4> t_resistanceArray, Buffer<float> velocityBufferX, Buffer<float> velocityBufferY)
	{
		const int2 cell{ getGlobalIndex2D() };

		if (isOutside(cell))
		{
			return;
		}

		const int width{ 2 * (c_parameters.gridSize.x / 2 + 1) };
		const int cellIndex{ cell.x + cell.y * width };

		const float2 velocity = t_windArray.read(cell);
		velocityBufferX[cellIndex] = velocity.x;
		velocityBufferY[cellIndex] = velocity.y;
	}

	__global__ void fftProjection(Buffer<hipComplex> frequencyBufferX, Buffer<hipComplex> frequencyBufferY)
	{
		const int2 cell{ getGlobalIndex2D() };
		const int2 size{ c_parameters.gridSize.x / 2 + 1, c_parameters.gridSize.y };

		if (isOutside(cell, size))
		{
			return;
		}

		const int cellIndex{ getCellIndex(cell, size) };

		hipComplex xterm{ frequencyBufferX[cellIndex] };
		hipComplex yterm{ frequencyBufferY[cellIndex] };

		const int iix{ cell.x };
		const int iiy{ cell.y > size.y / 2 ? cell.y - size.y : cell.y };

		const float kk{ static_cast<float>(iix * iix + iiy * iiy) };

		constexpr float viscosity{ 0.0f };
		const float diffusion = 1.0f / (1.0f + kk * viscosity * c_parameters.deltaTime);
		xterm.x *= diffusion;
		xterm.y *= diffusion;
		yterm.x *= diffusion;
		yterm.y *= diffusion;

		if (kk > 0.0f)
		{
			const float rkk{ 1.0f / kk };
			const float rkp{ iix * xterm.x + iiy * yterm.x };
			const float ikp{ iix * xterm.y + iiy * yterm.y };

			xterm.x -= rkk * rkp * iix;
			xterm.y -= rkk * ikp * iix;
			yterm.x -= rkk * rkp * iiy;
			yterm.y -= rkk * ikp * iiy;
		}

		frequencyBufferX[cellIndex] = xterm;
		frequencyBufferY[cellIndex] = yterm;
	}

	__global__ void finalizeProjection(Array2D<float2> t_windArray, Buffer<float> velocityBufferX, Buffer<float> velocityBufferY)
	{
		const int2 cell{ getGlobalIndex2D() };

		if (isOutside(cell))
		{
			return;
		}

		const int width{ 2 * (c_parameters.gridSize.x / 2 + 1) };

		const int cellIndex{ cell.x + cell.y * width };
		const float scale{ 1.0f / static_cast<float>(c_parameters.gridSize.x * c_parameters.gridSize.y) };

		const float2 velocity{ velocityBufferX[cellIndex], velocityBufferY[cellIndex] };
		t_windArray.write(cell, scale * velocity);
	}

	// Debug Operators for divergence reduction
	struct Unary
	{
		__device__ float operator()(float x)
		{
			return fabsf(x);
		}
	};
	struct Binary
	{
		__device__ float operator()(float x, float y)
		{
			return x + y;
		}
	};

	void pressureProjection(const LaunchParameters& t_launchParameters, const SimulationParameters& t_simulationParameters) 
	{
		Buffer<float> divergenceBuffer{ t_launchParameters.tmpBuffer + 2 * t_simulationParameters.cellCount };
		Buffer<float> pressureABuffer{ t_launchParameters.tmpBuffer + 0 * t_simulationParameters.cellCount };
		Buffer<float> pressureBBuffer{ t_launchParameters.tmpBuffer + 1 * t_simulationParameters.cellCount };

		if (t_launchParameters.projection.mode == ProjectionMode::Jacobi)
		{
			//multiplyWindShadowKernel<<<t_launchParameters.gridSize2D, t_launchParameters.blockSize2D>>>(t_launchParameters.windArray, t_launchParameters.resistanceArray);
			initDivergencePressureKernel<<<t_launchParameters.gridSize2D, t_launchParameters.blockSize2D >> > (t_launchParameters.windArray, divergenceBuffer, pressureABuffer);

			// Debug
			float div = thrust::transform_reduce(thrust::device, divergenceBuffer, divergenceBuffer + t_simulationParameters.cellCount, Unary(), 0.0f, Binary());
			printf("%f -> ", div / t_simulationParameters.cellCount);

			for (int i = 0; i < t_launchParameters.projection.jacobiIterations; ++i) 
			{
		        projectKernel<<<t_launchParameters.gridSize2D, t_launchParameters.blockSize2D>>>(t_launchParameters.resistanceArray, divergenceBuffer, pressureABuffer, pressureBBuffer);
		        std::swap(pressureABuffer, pressureBBuffer);
		    }

		    finalizeVelocities<<<t_launchParameters.gridSize2D, t_launchParameters.blockSize2D>>>(t_launchParameters.resistanceArray, t_launchParameters.windArray, pressureABuffer);
		 	
			// Debug
		    initDivergencePressureKernel<<<t_launchParameters.gridSize2D, t_launchParameters.blockSize2D>>>(t_launchParameters.windArray, divergenceBuffer, pressureABuffer);
		    div = thrust::transform_reduce(thrust::device, divergenceBuffer, divergenceBuffer + t_simulationParameters.cellCount, Unary(), 0.0f, Binary());
		    printf("%f\n", div / t_simulationParameters.cellCount);
		}
		else if (t_launchParameters.projection.mode == ProjectionMode::FFT)
		{
			//multiplyWindShadowKernel<<<t_launchParameters.gridSize2D, t_launchParameters.blockSize2D>>>(t_launchParameters.windArray, t_launchParameters.resistanceArray);
			
			// Debug
			initDivergencePressureKernel<<<t_launchParameters.gridSize2D, t_launchParameters.blockSize2D>>>(t_launchParameters.windArray, divergenceBuffer, pressureABuffer);
            float div = thrust::transform_reduce(thrust::device, divergenceBuffer, divergenceBuffer + t_simulationParameters.cellCount, Unary(), 0.0f, Binary());
            printf("%f -> ", div / t_simulationParameters.cellCount);

			setupProjection<<<t_launchParameters.gridSize2D, t_launchParameters.blockSize2D>>>(t_launchParameters.windArray, t_launchParameters.resistanceArray, t_launchParameters.projection.velocities[0], t_launchParameters.projection.velocities[1]);

		    CUFFT_CHECK_ERROR(hipfftExecR2C(t_launchParameters.projection.planR2C, (hipfftReal*)t_launchParameters.projection.velocities[0], (hipComplex*)t_launchParameters.projection.velocities[0]));
		    CUFFT_CHECK_ERROR(hipfftExecR2C(t_launchParameters.projection.planR2C, (hipfftReal*)t_launchParameters.projection.velocities[1], (hipComplex*)t_launchParameters.projection.velocities[1]));

			dim3 gridSize;
			gridSize.x = static_cast<unsigned int>(ceilf(static_cast<float>(t_simulationParameters.gridSize.x / 2 + 1) / 8.0f));
			gridSize.y = static_cast<unsigned int>(ceilf(static_cast<float>(t_simulationParameters.gridSize.y) / 8.0f));
			gridSize.z = 1;

		    fftProjection<<<gridSize, dim3{ 8, 8, 1 } >> >((hipComplex*)t_launchParameters.projection.velocities[0], (hipComplex*)t_launchParameters.projection.velocities[1]);

		    CUFFT_CHECK_ERROR(hipfftExecC2R(t_launchParameters.projection.planC2R, (hipComplex*)t_launchParameters.projection.velocities[0], (hipfftReal*)t_launchParameters.projection.velocities[0]));
		    CUFFT_CHECK_ERROR(hipfftExecC2R(t_launchParameters.projection.planC2R, (hipComplex*)t_launchParameters.projection.velocities[1], (hipfftReal*)t_launchParameters.projection.velocities[1]));

		    finalizeProjection<<<t_launchParameters.gridSize2D, t_launchParameters.blockSize2D>>>(t_launchParameters.windArray, t_launchParameters.projection.velocities[0], t_launchParameters.projection.velocities[1]);
		
			// Debug
		    initDivergencePressureKernel<<<t_launchParameters.gridSize2D, t_launchParameters.blockSize2D>>>(t_launchParameters.windArray, divergenceBuffer, pressureABuffer);
		    div = thrust::transform_reduce(thrust::device, divergenceBuffer, divergenceBuffer + t_simulationParameters.cellCount, Unary(), 0.0f, Binary());
		    printf("%f\n", div / t_simulationParameters.cellCount);
		}
	}
}