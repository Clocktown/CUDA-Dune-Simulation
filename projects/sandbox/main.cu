#include "hip/hip_runtime.h"
#include <sthe/cu/buffer.hpp>
#include <sthe/cu/array2d.hpp>
#include <sthe/cu/stopwatch.hpp>
#include <sthe/device/buffer.cuh>
#include <sthe/device/array2d.cuh>
#include <sthe/device/vector_extension.cuh>
#include <>

#define THREADS_1D 512
#define THREADS_2D 8
#define WIDTH 4096
#define HEIGHT 4096
#define COUNT (WIDTH * HEIGHT)
#define ITERS 1000

using namespace sthe;

__global__ void gaussBuffer1D(const device::Buffer<float4> t_in, device::Buffer<float4> t_out)
{
	const int idx{ static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x) };
	const int x{ idx % WIDTH };
	const int y{ idx / WIDTH };

	float4 sum{ make_float4(0.0f) };

	for (int i{ -1 }; i <= 1; ++i)
	{
		for (int j{ -1 }; j <= 1; ++j)
		{
			const int2 cell{ (x + j), (y + i) };

			if (cell.x < 0 || cell.y < 0 || cell.x >= WIDTH || cell.y >= HEIGHT)
			{
				continue;
			}

			sum += t_in[cell.x + WIDTH * cell.y];
		}
	}

	sum /= 9.0f;

	t_out[x + WIDTH * y] = sum;
}

__global__ void gaussBuffer2D(const device::Buffer<float4> t_in, device::Buffer<float4> t_out)
{
	const int x{ static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x) };
	const int y{ static_cast<int>(threadIdx.y + blockIdx.y * blockDim.y) };

	float4 sum{ make_float4(0.0f) };

	for (int i{ -1 }; i <= 1; ++i)
	{
		for (int j{ -1 }; j <= 1; ++j)
		{
			const int2 cell{ (x + j), (y + i) };

			if (cell.x < 0 || cell.y < 0 || cell.x >= WIDTH || cell.y >= HEIGHT)
			{
				continue;
			}

			sum += t_in[cell.x + WIDTH * cell.y];
		}
	}

	sum /= 9.0f;

	t_out[x + WIDTH * y] = sum;
}

__global__ void gaussArray2D(const device::Array2D<float4> t_in, device::Array2D<float4> t_out)
{
	const int2 idx{ static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x),
	                static_cast<int>(threadIdx.y + blockIdx.y * blockDim.y) };

	float4 sum{ make_float4(0.0f) };

	for (int i{ -1 }; i <= 1; ++i)
	{
		for (int j{ -1 }; j <= 1; ++j)
		{
			const int2 cell{ idx + make_int2(j, i) };

			sum += t_in.read(cell, hipBoundaryModeZero);
			//sum += t_in.sample(make_float2(cell) + 0.5f);
		}
	}

	sum /= 9.0f;

	t_out.write(idx, sum);
}

void main()
{
	hipTextureDesc tex{};
	tex.filterMode = hipFilterModePoint;
	tex.addressMode[0] = hipAddressModeBorder;
	tex.addressMode[1] = hipAddressModeBorder;
	tex.normalizedCoords = 0;

	cu::Buffer buffer1(COUNT, sizeof(float4));
	cu::Buffer buffer2(COUNT, sizeof(float4));
	cu::Array2D array1(WIDTH, HEIGHT, hipCreateChannelDesc<float4>());
	cu::Array2D array2(WIDTH, HEIGHT, hipCreateChannelDesc<float4>());
	device::Array2D<float4> deviceArray1;
	device::Array2D<float4> deviceArray2;
	deviceArray1.texture = array1.recreateTexture(tex);
	deviceArray1.surface = array1.recreateSurface();
	deviceArray2.texture = array2.recreateTexture(tex);
	deviceArray2.surface = array2.recreateSurface();

	const unsigned int blocks1D{ COUNT / THREADS_1D };
	const dim3 blocks2D{ WIDTH / THREADS_2D, HEIGHT / THREADS_2D };

	cu::Stopwatch sw;
	float timeBuffer1D{ 0.0f };
	float timeBuffer2D{ 0.0f };
	float timeArray2D{ 0.0f };

	for (int i{ 0 }; i < 100; ++i)
	{
		gaussBuffer1D<<<blocks1D, THREADS_1D>>> (buffer1.getData<float4>(), buffer2.getData<float4>());
		gaussBuffer2D<<<blocks2D, dim3{ THREADS_2D, THREADS_2D }>>>(buffer1.getData<float4>(), buffer2.getData<float4>());
		gaussArray2D<<<blocks2D, dim3{ THREADS_2D, THREADS_2D }>>>(deviceArray1, deviceArray2);
	}

	for (int i{ 0 }; i < ITERS; ++i)
	{
		sw.start();
		gaussBuffer1D<<<blocks1D, THREADS_1D>>>(buffer1.getData<float4>(), buffer2.getData<float4>());
		sw.stop();
		timeBuffer1D += sw.getTime();

		sw.start();
		gaussBuffer2D<<<blocks2D, dim3{ THREADS_2D, THREADS_2D }>>>(buffer1.getData<float4>(), buffer2.getData<float4>());
		sw.stop();
		timeBuffer2D += sw.getTime();

		sw.start();
		gaussArray2D<<<blocks2D, dim3{ THREADS_2D, THREADS_2D }>>>(deviceArray1, deviceArray2);
		sw.stop();
		timeArray2D += sw.getTime();
	}

	printf("Time Buffer 1D: %f\n", timeBuffer1D / ITERS);
	printf("Time Buffer 2D: %f\n", timeBuffer2D / ITERS);
	printf("Time Array 2D:  %f\n", timeArray2D / ITERS);
}
